#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * \file layer_norm.cu
 * \brief Implements Ba et. al, Layer Normalization (https://arxiv.org/abs/1607.06450).
*/
#include "./layer_norm-inl.h"

using namespace mshadow::cuda;

namespace mxnet {
namespace op {

template <typename DType>
__device__ __forceinline__ DType WARP_SHFL(DType value, int src_lane,
                                           int width = 32, unsigned int mask = 0xffffffff)
{
#if CUDA_VERSION >= 9000
  return __shfl_sync(mask, value, src_lane, width);
#else
  return __shfl(value, src_lane, width);
#endif
}

template<typename DType>
__device__ __forceinline__ DType rsqrt(DType v) {
  return DType(1) / sqrt(v);
}

template<>
__device__ __forceinline__ float rsqrt(float v) {
  return rsqrtf(v);
}

template<>
__device__ __forceinline__ double rsqrt(double v) {
  return rsqrt(v);
}


/* A single updating step of the Welford's online algorithm to calculate the mean and variance.
 * The value 'curr' will be accumulated to the (mean, sigma2, count) triplet.
 *
 */
template<typename DType>
__device__ __forceinline__ void welford_online_sum_step(const DType curr,
                                                        DType& mean,
                                                        DType& sigma2,
                                                        DType& count) {
  count += DType(1);
  DType delta = curr - mean;
  mean += delta / count;
  sigma2 += delta * (curr - mean);
}

/* Merge the mean/variance of two partitions. It's the key step of the Chan's parallel algorithm.
 * The (lhs_mean, lhs_sigma2, lhs_count) will be merged into (rhs_mean, rhs_sigma2, rhs_count)
 *
 * See https://en.wikipedia.org/wiki/Algorithms_for_calculating_variance for more details.
 *
 *  TODO(sxjscience) Explore the possibility of int lhs_count and rhs_count
 */
template<typename DType>
__device__ __inline__ void chan_merge_partition(const DType lhs_mean,
                                                const DType lhs_sigma2,
                                                const DType lhs_count,
                                                DType& rhs_mean,
                                                DType& rhs_sigma2,
                                                DType& rhs_count) {
  DType delta = rhs_mean - lhs_mean;
  DType nA = lhs_count;
  DType nB = rhs_count;
  rhs_count = nA + nB;
  if (rhs_count > DType(0)) {
    nA = nA / rhs_count;
    nB = nB / rhs_count;
    rhs_mean = nA * lhs_mean + nB * rhs_mean;
    rhs_sigma2 = rhs_sigma2 + lhs_sigma2 + delta * delta * nA * nB * rhs_count;
  } else {
    rhs_mean = DType(0);
    rhs_sigma2 = DType(0);
  }
}


template<typename DType>
__device__ __forceinline__ void _block_welford_online_sum(const int tid,
                                                          const int nthread,
                                                          const DType* __restrict__ col_vals,
                                                          const int nchannel,
                                                          DType& mean,
                                                          DType& sigma2,
                                                          DType& count) {
  int l = 4 * tid;
  for (; l + 3 < nchannel; l += 4 * nthread) {
    for (int i = 0; i < 4; ++i) {
      welford_online_sum_step(col_vals[l + i], mean, sigma2, count);
    }
  }
  for(; l < nchannel; ++l) {
    welford_online_sum_step(col_vals[l], mean, sigma2, count);
  }
}

template<>
__device__ __forceinline__ void _block_welford_online_sum(const int tid,
                                                          const int nthread,
                                                          const float* __restrict__ col_vals,
                                                          const int nchannel,
                                                          float& mean,
                                                          float& sigma2,
                                                          float& count) {
  int l = 4 * tid;
  const float4* col_vals_float4 = reinterpret_cast<const float4*>(col_vals);
  for (; l + 3 < nchannel; l += 4 * nthread) {
    float4 vec_vals = col_vals_float4[l];
    welford_online_sum_step(vec_vals.x, mean, sigma2, count);
    welford_online_sum_step(vec_vals.y, mean, sigma2, count);
    welford_online_sum_step(vec_vals.z, mean, sigma2, count);
    welford_online_sum_step(vec_vals.w, mean, sigma2, count);
  }
  for(; l < nchannel; ++l) {
    welford_online_sum_step(col_vals[l], mean, sigma2, count);
  }
}


/* Fused CUDA kernel for layer normalization. It computes the LayerNorm when axis=-1.
 * Shape of the input tensors:
 *      in_data = (nbatch, nchannel)
 *      gamma = (nchannel,)
 *      beta = (nchannel,)
 *      out_data = (nchannel,)
 *      mean_data = (nbatch,)
 *      var_data = (nbatch,)
 *  It's always launched with (blockDim.x, blockDim.y) = (WARP_SIZE, blockDim.y)
 *  Also, when blockDim.y > 1, it requires shared memory that has size:
 *      sizeof(DType) * blockDim.y + sizeof(DType) * blockDim.y / 2
 */
template<typename DType>
__global__ void LayerNormFusedForwardKernelContig(const int nbatch,
                                                  const int nchannel,
                                                  const DType eps,
                                                  const DType* __restrict__ in_data,
                                                  const DType* __restrict__ gamma,
                                                  const DType* __restrict__ beta,
                                                  DType* __restrict__ out_data,
                                                  DType* __restrict__ mean_data,
                                                  DType* __restrict__ std_data) {
  int bid = blockIdx.x + blockIdx.y * gridDim.x;
  const int nthread = blockDim.x * blockDim.y;
  DType count = 0;
  DType mean = 0;
  DType sigma2 = 0;
  const int N_ACCUM = 4;  // TODO(sxjscience) Profile
  extern __shared__ char buf[];  // Shared memory size

  if (bid < nbatch) {
    int tid = threadIdx.x + threadIdx.y * blockDim.x;
    const DType* col_vals = in_data + bid * nchannel;
    // Each thread takes charge of 4 consecutive numbers
    // To minimize branch divergence, we split the for-loop into two parts.
    _block_welford_online_sum(tid, nthread, col_vals, nchannel, mean, sigma2, count);

    // Merge the mean/sigma2 within a warp
    // Use the Chan's Parallel Algorithm to merge all (mean, sigma2, counts)
    // within a warp of threads.
    // After calling the function, threadIdx.x == 0 will store the result of
    // the aggregated (mean, sigma2, counts).
    for (int l = 0; l <= 4; ++l) {
      int src_lane = (threadIdx.x + (1<<l)) & 31;
      DType meanB = WARP_SHFL(mean, src_lane);
      DType sigma2B = WARP_SHFL(sigma2, src_lane);
      DType countB = WARP_SHFL(count, src_lane);
      chan_merge_partition(meanB, sigma2B, countB, mean, sigma2, count);
    }
    if (blockDim.y == 1) {
      mean = WARP_SHFL(mean, 0);
      sigma2 = WARP_SHFL(sigma2 / nchannel, 0); // Calculate the variance
    } else {
      // Inter-warp reduction. Copy the upper-half of the warps to shared memory
      // and merge with the lower-half warp
      DType* mean_buf = reinterpret_cast<DType*>(buf);
      DType* sigma2_buf = reinterpret_cast<DType*>(buf + sizeof(DType) * blockDim.y / 2);
      DType* count_buf = reinterpret_cast<DType*>(buf + sizeof(DType) * blockDim.y);
      for (int offset = blockDim.y / 2; offset > 0; offset /= 2) {
        if (threadIdx.x == 0 && threadIdx.y >= offset && threadIdx.y < 2 * offset) {
          const int idx = threadIdx.y - offset;
          mean_buf[idx] = mean;
          sigma2_buf[idx] = sigma2;
          count_buf[idx] = count;
        }
        __syncthreads();
        if (threadIdx.x == 0 && threadIdx.y < offset) {
          chan_merge_partition(mean_buf[threadIdx.y],
                               sigma2_buf[threadIdx.y],
                               count_buf[threadIdx.y], mean, sigma2, count);
        }
        __syncthreads();
      }
      // Broadcast the result to all threads
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        mean_buf[0] = mean;
        sigma2_buf[0] = sigma2;
      }
      __syncthreads();
      mean = mean_buf[0];
      sigma2 = sigma2_buf[0] / nchannel;
    }
    // Calculate the out_data: gamma * (x - mean) / sqrt(var + eps) + beta
    DType std_eps = sqrt(sigma2 + eps);
    DType invstd_eps = DType(1.0) / std_eps;
    DType* out_col_val = out_data + bid * nchannel;

    if (gamma != NULL && beta != NULL) {
      for (int i = tid; i < nchannel; i += nthread) {
        out_col_val[i] = gamma[i] * invstd_eps * (col_vals[i] - mean) + beta[i];
      }
    } else if (gamma == NULL && beta != NULL) {
      for (int i = tid; i < nchannel; i += nthread) {
        out_col_val[i] = invstd_eps * (col_vals[i] - mean) + beta[i];
      }
    } else if (gamma != NULL && beta == NULL) {
      for (int i = tid; i < nchannel; i += nthread) {
        out_col_val[i] = gamma[i] * invstd_eps * (col_vals[i] - mean);
      }
    } else {
      for (int i = tid; i < nchannel; i += nthread) {
        out_col_val[i] = invstd_eps * (col_vals[i] - mean);
      }
    }
    // Write the out_data and var_data
    if(threadIdx.x == 0 && threadIdx.y == 0) {
      mean_data[bid] = mean;
      std_data[bid] = std_eps;
    }
  }
}

void LayerNormGPUContig(const LayerNormParam param,
                        const OpContext& ctx, const std::vector<TBlob>& inputs,
                        const std::vector<OpReqType>& req,
                        const std::vector<TBlob>& outputs) {
  using namespace mshadow;
  CHECK_EQ(inputs.size(), 3U);
  mxnet::TShape data_shape(2);
  mxnet::TShape mean_shape(1);
  size_t in_ndim = inputs[layernorm::kData].ndim();
  data_shape[0] = mean_shape[0] = inputs[layernorm::kData].shape_.ProdShape(0, in_ndim - 1);
  data_shape[1] = inputs[layernorm::kData].shape_[in_ndim - 1];
  const TBlob in_data = inputs[layernorm::kData].reshape(data_shape);
  const TBlob gamma = inputs[layernorm::kGamma];
  const TBlob beta = inputs[layernorm::kBeta];
  const TBlob out_data = outputs[layernorm::kOut].reshape(data_shape);
  const TBlob mean_data = outputs[layernorm::kMean].reshape(mean_shape);
  const TBlob std_data = outputs[layernorm::kStd].reshape(mean_shape);
  // Make sure the inputs are contiguous
  CHECK_EQ(in_data.CheckContiguous(), true);
  CHECK_EQ(gamma.CheckContiguous(), true);
  CHECK_EQ(beta.CheckContiguous(), true);
  CHECK_EQ(out_data.CheckContiguous(), true);
  CHECK_EQ(mean_data.CheckContiguous(), true);
  CHECK_EQ(std_data.CheckContiguous(), true);

  // Lauch the kernel. The dynamic shared memory size is sizeof(DType) * threadDim.y + sizeof(DType) *
  int nbatch = data_shape[0];
  int nchannel = data_shape[1];
  float eps = param.eps;
  int ngrid_x = (nbatch > kMaxGridDim) ? (nbatch + kBaseGridNum - 1) / kBaseGridNum : nbatch;
  int ngrid_y = (nbatch > kMaxGridDim) ? kBaseGridNum : 1;
  int nthread_y = 0;
  const dim3 dimGrid(ngrid_x, ngrid_y, 1);
  if(nchannel <= 32) {
    nthread_y = 1;
  } else if(nchannel <= 64) {
    nthread_y = 2;
  } else {
    nthread_y = 4;
  }
  hipStream_t stream = Stream<gpu>::GetStream(ctx.get_stream<gpu>());
  const dim3 dimBlock(32, nthread_y, 1);
  MSHADOW_REAL_TYPE_SWITCH(in_data.type_flag_, DType, {
    int nshared = nthread_y > 1 ? nthread_y * sizeof(DType) + (nthread_y / 2) * sizeof(DType) : 0;
    CheckLaunchParam(dimGrid, dimBlock);
    LayerNormFusedForwardKernelContig<<<dimGrid, dimBlock, nshared, stream>>>
     (nbatch, nchannel, static_cast<DType>(eps),
      in_data.dptr<DType>(), gamma.dptr<DType>(), beta.dptr<DType>(),
      out_data.dptr<DType>(), mean_data.dptr<DType>(), std_data.dptr<DType>());
    MSHADOW_CUDA_POST_KERNEL_CHECK(LayerNormFusedForwardKernelContig);
  });
}

template<>
void LayerNormCompute<gpu>(const nnvm::NodeAttrs& attrs,
                           const OpContext& ctx, const std::vector<TBlob>& inputs,
                           const std::vector<OpReqType>& req,
                           const std::vector<TBlob>& outputs) {
  const LayerNormParam& param = nnvm::get<LayerNormParam>(attrs.parsed);
  if (req[0] == kNullOp) return;
  CHECK_NE(req[0], kAddTo);
  int axis = param.axis;
  if (axis < 0) {
    axis += static_cast<int>(inputs[0].ndim());
  }
  CHECK(axis >= 0 && axis < inputs[0].ndim()) << "Channel axis out of range: " << param.axis;
  if(axis == inputs[0].ndim() - 1) {
    // Try to use the accelerated CUDA kernels
    return LayerNormGPUContig(param, ctx, inputs, req, outputs);
  }
  return LayerNormComputeGeneral<gpu>(attrs, ctx, inputs, req, outputs);
}

NNVM_REGISTER_OP(LayerNorm)
.set_attr<FCompute>("FCompute<gpu>", LayerNormCompute<gpu>);

NNVM_REGISTER_OP(_backward_LayerNorm)
.set_attr<FCompute>("FCompute<gpu>", LayerNormGradCompute<gpu>);

}  // namespace op
}  // namespace mxnet
