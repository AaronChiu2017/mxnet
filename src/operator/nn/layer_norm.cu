#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * Copyright (c) 2015 by Contributors
 * \file layer_norm.cu
 * \brief Implements Ba et. al, Layer Normalization (https://arxiv.org/abs/1607.06450).
*/
#include "./layer_norm-inl.h"

namespace mxnet {
namespace op {


void LayerNormCompute(const nnvm::NodeAttrs& attrs,
                      const OpContext& ctx, const std::vector<TBlob>& inputs,
                      const std::vector<OpReqType>& req,
                      const std::vector<TBlob>& outputs) {
  return LayerNormComputeGeneral<gpu>(attrs, ctx, inputs, req, outputs);
}

NNVM_REGISTER_OP(LayerNorm)
.set_attr<FCompute>("FCompute<gpu>", LayerNormCompute);

NNVM_REGISTER_OP(_backward_LayerNorm)
.set_attr<FCompute>("FCompute<gpu>", LayerNormGradCompute<gpu>);

}  // namespace op
}  // namespace mxnet
