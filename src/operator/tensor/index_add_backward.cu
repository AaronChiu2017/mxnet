#include "hip/hip_runtime.h"
/*
 * Licensed to the Apache Software Foundation (ASF) under one
 * or more contributor license agreements.  See the NOTICE file
 * distributed with this work for additional information
 * regarding copyright ownership.  The ASF licenses this file
 * to you under the Apache License, Version 2.0 (the
 * "License"); you may not use this file except in compliance
 * with the License.  You may obtain a copy of the License at
 *
 *   http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing,
 * software distributed under the License is distributed on an
 * "AS IS" BASIS, WITHOUT WARRANTIES OR CONDITIONS OF ANY
 * KIND, either express or implied.  See the License for the
 * specific language governing permissions and limitations
 * under the License.
 */

/*!
 * \file index_add.cu
 * \brief GPU implementation of index_add operator
 */

#include <hipcub/hipcub.hpp>
#include "./index_add-inl.h"
#include "../tensor/util/tensor_util-inl.cuh"
#include "../tensor/util/tensor_util-inl.h"

namespace mxnet {
namespace op {

template<typename xpu, typename DType>
void IndexAddOpBackwardACalc(mshadow::Stream<xpu> *s,
                             DType* grad_a, const DType* ograd,
                             const mshadow::Shape<MXNET_SPECIAL_MAX_NDIM>& stride,
                             const int tail_size, const int ind_num,
                             const int ind_ndim, const int32_t* ind_vec,
                             const int req, const int out_ndim) {
  using namespace mxnet_op;
  using namespace mshadow;
  Kernel<IndexAddBackwardAKernel<DType>, xpu>::Launch(
    s, ind_num, grad_a, ograd, stride, tail_size,
    ind_num, ind_ndim, ind_vec, req, out_ndim);
}

template<typename DType>
struct IndexAddBackwardValGPUKernel {
  MSHADOW_XINLINE static void Map(size_t i, DType* grad_val,
                                  const DType* ograd,
                                  const mshadow::Shape<MXNET_SPECIAL_MAX_NDIM>& ograd_tail_shape,
                                  const mshadow::Shape<MXNET_SPECIAL_MAX_NDIM>& ograd_pre_stride,
                                  const mshadow::Shape<MXNET_SPECIAL_MAX_NDIM>& val_stride,
                                  const mshadow::Shape<MXNET_SPECIAL_MAX_NDIM>& val_shape,
                                  const int ograd_tail_size, const int ind_num,
                                  const int ind_ndim, const int32_t* ind_vec,
                                  const int out_ndim) {
    size_t id = 0;
    int seg = MXNET_SPECIAL_MAX_NDIM - out_ndim;
    for (int dim = 0; dim < ind_ndim; ++dim) {
      id += ograd_pre_stride[seg + dim] * ind_vec[dim * ind_num + i];
    }
    id *= ograd_tail_size;
    for (int _i = 0; _i < ograd_tail_size; ++_i) {
      mshadow::Shape<MXNET_SPECIAL_MAX_NDIM> ograd_tail_id =
        mxnet_op::unravel(_i, ograd_tail_shape);
      mshadow::Shape<MXNET_SPECIAL_MAX_NDIM> val_id;
      for (int _j = seg; _j < seg + out_ndim; ++_j) {
        val_id[_j] = (val_shape[_j] == 1) ? 0 : ograd_tail_id[_j];
      }
      val_id[seg + ind_ndim - 1] = (val_shape[seg + ind_ndim - 1] == 1) ? 0 : i;
      index_t val_dest = mxnet_op::dot(val_id, val_stride);
      atomicAdd(&grad_val[val_dest], ograd[id + _i]);
    }
  }
};

template<typename xpu, typename DType>
void IndexAddOpBackwardValCalc(mshadow::Stream<xpu> *s,
                               DType* grad_val, const DType* ograd,
                               const mshadow::Shape<MXNET_SPECIAL_MAX_NDIM>& ograd_tail_shape,
                               const mshadow::Shape<MXNET_SPECIAL_MAX_NDIM>& ograd_pre_stride,
                               const mshadow::Shape<MXNET_SPECIAL_MAX_NDIM>& val_stride,
                               const mshadow::Shape<MXNET_SPECIAL_MAX_NDIM>& val_shape,
                               const int tail_size, const int ind_num,
                               const int ind_ndim, const int32_t* ind_vec,
                               const int out_ndim) {
  using namespace mxnet_op;
  using namespace mshadow;
  Kernel<IndexAddBackwardValGPUKernel<DType>, xpu>::Launch(
    s, ind_num, grad_val, ograd, ograd_tail_shape, ograd_pre_stride,
    val_stride, val_shape, tail_size, ind_num, ind_ndim, ind_vec, out_ndim);
}

NNVM_REGISTER_OP(_backward_index_add)
.set_attr<FCompute>("FCompute<gpu>", IndexAddOpBackward<gpu>);

}  // namespace op
}  // namespace mxnet

