#include "hip/hip_runtime.h"
/*!
 *  Copyright (c) 2015 by Contributors
 * \file circ_conv.cu
 * \brief GPU Implementation of circular convolution
 */
// this will be invoked by gcc and compile GPU version
#include "./circ_conv-inl.h"
#include "mshadow/cuda/tensor_gpu-inl.cuh"
#define MX_CUDA_CHECK(condition) \
  /* Code block avoids redefinition of hipError_t error */ \
  do { \
    hipError_t error = condition; \
    CHECK_EQ(error, hipSuccess) << " " << hipGetErrorString(error); \
    } while (0)


namespace mshadow {
namespace cuda {

template<typename DType>
DType uppow2(DType num) {
  return 1 << static_cast<DType>(ceil(log2(num)));
}

template<typename DType>
__device__ DType modn(DType a, DType N) {
  return (a < 0) ? (a + N) : ((a >= N) ? a - N : a);
}

template<int x_bits, typename DType>
__global__ void CircularConvolution1DForwardKernel(const int content_size,
                                                   const int kernel_size,
                                                   const DType *data,
                                                   const DType *weight,
                                                   DType *out) {
  const unsigned x_size = 1 << x_bits;
  const index_t y = blockIdx.x;
  const int tid = static_cast<int>(threadIdx.x);
  __shared__ DType s_data_[x_size];
  __shared__ DType s_weight_[x_size];
  if (tid < content_size) {
    s_data_[tid] = data[y * content_size + tid];
  } else {
    s_data_[tid] = 0;
  }
  if (tid < kernel_size) {
    s_weight_[tid] = weight[y * kernel_size + tid];
  } else {
    s_weight_[tid] = 0;
  }
  __syncthreads();
  if (tid < content_size) {
    for (int i = 0; i < kernel_size; ++i) {
      int indx = modn(tid - i, content_size);
      out[y * content_size + tid] += s_data_[indx] * s_weight_[i];
    }
  }
}

template<int x_bits, typename DType>
__global__ void CircularConvolution1DBackwardKernel(const int content_size,
                                                    const int kernel_size,
                                                    const DType *out_grad,
                                                    const DType *data,
                                                    const DType *weight,
                                                    DType *data_grad,
                                                    DType *weight_grad) {
  const unsigned x_size = 1 << x_bits;
  const index_t y = blockIdx.x;
  const int tid = static_cast<int>(threadIdx.x);
  __shared__ DType s_data_[x_size];
  __shared__ DType s_weight_[x_size];
  __shared__ DType s_out_grad_[x_size];
  __shared__ DType s_red_[x_size];
  if (tid < content_size) {
    s_data_[tid] = data[y * content_size + tid];
    s_out_grad_[tid] = out_grad[y * content_size + tid];
    s_red_[tid] = 0;
  } else {
    s_data_[tid] = 0;
    s_out_grad_[tid] = 0;
    s_red_[tid] = 0;
  }
  if (tid < kernel_size) {
    s_weight_[tid] = weight[y * kernel_size + tid];
  } else {
    s_weight_[tid] = 0;
  }
  __syncthreads();
  if (tid < content_size) {
    for (int i = 0; i < kernel_size; ++i) {
      int indx = modn(tid + i, content_size);
      data_grad[y * content_size + tid] += s_out_grad_[indx] * s_weight_[i];
    }
  }
  for (int i = 0; i < kernel_size; ++i) {
    if (tid < content_size) {
      int indx = modn(tid - i, content_size);
      s_red_[tid] = s_out_grad_[tid] * s_data_[indx];
    }
    __syncthreads();
    Reduce1D<red::sum, x_bits, DType>(s_red_);
    __syncthreads();
    if (tid == 0) {
      weight_grad[y * kernel_size + i] += s_red_[0];
    }
  }
}
}  // namespace cuda

template<typename DType>
inline void CircularConvolution1DForwardImpl_(const Tensor<gpu, 2, DType> &out,
                                              const Tensor<gpu, 2, DType> &data,
                                              const Tensor<gpu, 2, DType> &weight) {
  using namespace cuda;
  DType *out_ = out.dptr_;
  const DType *data_ = data.dptr_;
  const DType *weight_ = weight.dptr_;
  const int batch_size = data.size(0);
  const int content_size = data.size(1);
  const int kernel_size = weight.size(1);
  CHECK(kBaseThreadNum > content_size);
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid(batch_size);
  CheckLaunchParam(dimGrid, dimBlock, "Circular Convolution Forward");
  //TODO Optimize the kernel!
  hipStream_t stream = Stream<gpu>::GetStream(out.stream_);
  CircularConvolution1DForwardKernel<kBaseThreadBits, DType> << <dimGrid, dimBlock, 0, stream >> > (
    content_size, kernel_size, data_, weight_, out_);
  MX_CUDA_CHECK(hipPeekAtLastError());
  return;
}

template<typename DType>
inline void CircularConvolution1DBackwardImpl_(const Tensor<gpu, 2, DType> &out_grad,
                                               const Tensor<gpu, 2, DType> &data_grad,
                                               const Tensor<gpu, 2, DType> &weight_grad,
                                               const Tensor<gpu, 2, DType> &data,
                                               const Tensor<gpu, 2, DType> &weight) {
  using namespace cuda;
  const DType *out_grad_ = out_grad.dptr_;
  DType *data_grad_ = data_grad.dptr_;
  DType *weight_grad_ = weight_grad.dptr_;
  const DType *data_ = data.dptr_;
  const DType *weight_ = weight.dptr_;
  const int batch_size = data_grad.size(0);
  const int content_size = data_grad.size(1);
  const int kernel_size = weight_grad.size(1);
  CHECK(kBaseThreadNum > content_size);
  dim3 dimBlock(kBaseThreadNum);
  dim3 dimGrid(batch_size);
  CheckLaunchParam(dimGrid, dimBlock, "Circular Convolution Backward");
  //TODO Optimize the kernel!
  hipStream_t stream = Stream<gpu>::GetStream(out_grad.stream_);
  CircularConvolution1DBackwardKernel<kBaseThreadBits, DType> << <dimGrid, dimBlock, 0, stream >> > (
    content_size, kernel_size, out_grad_, data_, weight_, data_grad_, weight_grad_);
  MX_CUDA_CHECK(hipPeekAtLastError());
  return;
}
}  // namespace mshadow